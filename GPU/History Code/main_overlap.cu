//
// Created by Lenovo on 25-03-01.
//

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <fstream>
#include <cassert>
#include <filesystem>
#include <ctime>

namespace fs = std::filesystem;

// macro: constant parameters
#define NUM_VARS 4  // number of independent variables
#define nGhost 2  // number of ghost cells in each direction
#define C 0.8  // CFL number
#define gamma 1.4  // adiabatic index of ideal gas
#define nThreadsX 32  // number of threads per block in x-direction
#define nThreadsY 32  // number of threads per block in y-direction
#define nThreadsXSLICX 32  // number of threads per block in x-direction for SLIC evolution in x-direction
#define nThreadsYSLICX 4  // number of threads per block in y-direction for SLIC evolution in x-direction
#define nThreadsXSLICY 4  // number of threads per block in x-direction for SLIC evolution in y-direction
#define nThreadsYSLICY 32  // number of threads per block in y-direction for SLIC evolution in y-direction

// macro: debugging
#define CUDA_CHECK {\
    hipDeviceSynchronize();\
    hipError_t err = hipGetLastError();\
    if(err){\
        std::cout << "Error: " << hipGetErrorString(err) << " line " << __LINE__ << std::endl;\
        exit(1);\
    }\
}

enum Processor {CPU, GPU};

// structure: data storage and process
struct Grid {

    // attributes
    double* data;
    int nCellsX;
    int nCellsY;
    Processor processor;

    // format for building an instance
    Grid(const int nX, const int nY, Processor p) {
        nCellsX = nX;
        nCellsY = nY;
        processor = p;
        switch (processor) {
            case CPU:
                data = new double[(nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost) * NUM_VARS];
                break;
            case GPU:
                hipMalloc((void **)& data, (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost) * NUM_VARS * sizeof(double));
                break;
        }
    }

    // operator functions
    __device__ __host__
    double operator() (const int i, const int j, const int v) const {
        return data[i + j * (nCellsX + 2 * nGhost) + v * (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost)];
    }
    __device__ __host__
    double& operator() (const int i, const int j, const int v) {
        return data[i + j * (nCellsX + 2 * nGhost) + v * (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost)];
    }
};


// function: transform from primitive to conservative on CPU
void prim2consHost(double* u_ij_cons, const double* u_ij_prim) {

    const double rho = u_ij_prim[0];
    const double u = u_ij_prim[1];
    const double v = u_ij_prim[2];
    const double p = u_ij_prim[3];

    u_ij_cons[0] = rho;  // rho
    u_ij_cons[1] = rho * u;  // momx
    u_ij_cons[2] = rho * v;  // momy
    u_ij_cons[3] = p / (gamma - 1) + 0.5 * rho * (pow(u, 2) + pow(v, 2));  // E
}


// function: transform from primitive to conservative on GPU
__device__ void prim2consDevice(double* u_ij_cons, const double* u_ij_prim) {

    const double rho = u_ij_prim[0];
    const double u = u_ij_prim[1];
    const double v = u_ij_prim[2];
    const double p = u_ij_prim[3];

    u_ij_cons[0] = rho;  // rho
    u_ij_cons[1] = rho * u;  // momx
    u_ij_cons[2] = rho * v;  // momy
    u_ij_cons[3] = p / (gamma - 1) + 0.5 * rho * (pow(u, 2) + pow(v, 2));  // E
}


// function: transform from conservative to primitive on CPU
void cons2primHost(double* u_ij_prim, const double* u_ij_cons) {

    const double rho = u_ij_cons[0];
    const double momx = u_ij_cons[1];
    const double momy = u_ij_cons[2];
    const double E = u_ij_cons[3];

    u_ij_prim[0] = rho;  // rho
    u_ij_prim[1] = momx / rho;  // u
    u_ij_prim[2] = momy / rho;  // v
    u_ij_prim[3] = (gamma - 1) * (E - 0.5 * pow(momx, 2) / rho - 0.5 * pow(momy, 2) / rho);  // p
}


// function: transform from conservative to primitive on GPU
__device__ void cons2primDevice(double* u_ij_prim, const double* u_ij_cons) {

    const double rho = u_ij_cons[0];
    const double momx = u_ij_cons[1];
    const double momy = u_ij_cons[2];
    const double E = u_ij_cons[3];

    u_ij_prim[0] = rho;  // rho
    u_ij_prim[1] = momx / rho;  // u
    u_ij_prim[2] = momy / rho;  // v
    u_ij_prim[3] = (gamma - 1) * (E - 0.5 * pow(momx, 2) / rho - 0.5 * pow(momy, 2) / rho);  // p
}


// function: set transmissive boundary conditions
__global__ void setBoundaryCondition(Grid u) {

    const int nCellsX = u.nCellsX, nCellsY = u.nCellsY;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // transmissive boundary condition
    for (int v = 0; v < NUM_VARS; v++) {
        // lower boundary
        if (i >= nGhost && i < nCellsX + nGhost && j >= 0 && j < nGhost) {
            u(i, j, v) = u(i, nGhost, v);
        }
        // upper boundary
        if (i >= nGhost && i < nCellsX + nGhost && j >= nCellsY + nGhost && j < nCellsY + 2 * nGhost) {
            u(i, j, v) = u(i, nCellsY + nGhost - 1, v);
        }
        // left boundary
        if (j >= 0 && j < nCellsY + 2 * nGhost && i >= 0 && i < nGhost) {
            u(i, j, v) = u(nGhost, j, v);
        }
        // right boundary
        if (j >= 0 && j < nCellsY + 2 * nGhost && i >= nCellsX + nGhost && i < nCellsX + 2 * nGhost) {
            u(i, j, v) = u(nCellsX + nGhost - 1, j, v);
        }
    }
}


// function: calculate the maximum velocity in each block on GPU
__global__ void computeAmaxOpt(double* aDevice, Grid u) {

    // shared memory for current block
    __shared__ double aBlock[nThreadsX][nThreadsY];

    // variable substitution
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i_local = threadIdx.x, j_local = threadIdx.y;
    double cur_a = 0.0;

    // calculate a's in current block
    if (i >= nGhost && i < u.nCellsX + nGhost && j >= nGhost && j < u.nCellsY + nGhost) {
        double u_prim[NUM_VARS];
        double u_cons[NUM_VARS];
        for (int v = 0; v < NUM_VARS; v++) {
            u_cons[v] = u(i, j, v);
        }
        cons2primDevice(u_prim, u_cons);
        double cur_rho = u_prim[0], cur_vx = u_prim[1], cur_vy = u_prim[2], cur_p = u_prim[3];
        double cur_v = pow(pow(cur_vx, 2) + pow(cur_vy, 2), 0.5);
        double Cs = pow(gamma * cur_p / cur_rho, 0.5);
        cur_a = cur_v + Cs;
    }
    aBlock[threadIdx.x][threadIdx.y] = cur_a;
    __syncthreads();

    // block-wise reduction
    if (blockDim.x >= 32 && blockDim.y >= 32 && i_local < 16 && j_local < 16) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 16]),
            fmax(aBlock[i_local + 16][j_local], aBlock[i_local + 16][j_local + 16]));
        __syncthreads();
    }
    if (blockDim.x >= 16 && blockDim.y >= 16 && i_local < 8 && j_local < 8) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 8]),
            fmax(aBlock[i_local + 8][j_local], aBlock[i_local + 8][j_local + 8]));
        __syncthreads();
    }
    if (blockDim.x >= 8 && blockDim.y >= 8 && i_local < 4 && j_local < 4) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 4]),
            fmax(aBlock[i_local + 4][j_local], aBlock[i_local + 4][j_local + 4]));
        __syncthreads();
    }
    if (blockDim.x >= 4 && blockDim.y >= 4 && i_local < 2 && j_local < 2) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 2]),
            fmax(aBlock[i_local + 2][j_local], aBlock[i_local + 2][j_local + 2]));
        __syncthreads();
    }
    if (blockDim.x >= 2 && blockDim.y >= 2 && i_local < 1 && j_local < 1) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 1]),
            fmax(aBlock[i_local + 1][j_local], aBlock[i_local + 1][j_local + 1]));
        __syncthreads();
    }

    // a_max in current block
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        aDevice[blockIdx.y * gridDim.x + blockIdx.x] = aBlock[threadIdx.x][threadIdx.y];
    }
}


// function: calculate the velocity in each cell on GPU
__global__ void computeAmax(double* aDevice, Grid u) {

    // variable substitution
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    double cur_a = 0.0;

    // calculate a
    if (i >= nGhost && i < u.nCellsX + nGhost && j >= nGhost && j < u.nCellsY + nGhost) {
        double u_prim[NUM_VARS];
        double u_cons[NUM_VARS];
        for (int v = 0; v < NUM_VARS; v++) {
            u_cons[v] = u(i, j, v);
        }
        cons2primDevice(u_prim, u_cons);
        double cur_rho = u_prim[0], cur_vx = u_prim[1], cur_vy = u_prim[2], cur_p = u_prim[3];
        double cur_v = pow(pow(cur_vx, 2) + pow(cur_vy, 2), 0.5);
        double Cs = pow(gamma * cur_p / cur_rho, 0.5);
        cur_a = cur_v + Cs;
    }

    aDevice[j * blockDim.x * gridDim.x + i] = cur_a;
}


// function: calculate time step
double computeTimeStep(Grid u, const double& dx, const double& dy, const dim3& dimGrid, const dim3& dimBlock, bool optTime) {

    // calculate a's on GPU
    double* aDevice;
    int a_size = 0;
    if (optTime) {  // with shared memory optimization
        a_size = dimGrid.x * dimGrid.y;
        hipMalloc(&aDevice, a_size * sizeof(double));
        computeAmaxOpt<<<dimGrid, dimBlock>>>(aDevice, u);
        CUDA_CHECK;
    } else {  // without shared memory optimization
        a_size = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y;
        hipMalloc(&aDevice, a_size * sizeof(double));
        computeAmax<<<dimGrid, dimBlock>>>(aDevice, u);
        CUDA_CHECK;
    }

    // transfer data to CPU
    double* aHost = new double [a_size];
    hipMemcpy(aHost, aDevice, a_size * sizeof(double), hipMemcpyDeviceToHost);

    // for stability: numerical dependence stencil should contain the largest wave speed
    double a_max = 0.0;
    for (int i = 0; i < a_size; i++) {
        a_max = std::max(a_max, aHost[i]);
    }
    hipFree(aDevice);
    delete[] aHost;
    const double timeStep = C * std::min(dx, dy) / a_max;
    return timeStep;
}


// function: calculate slope limiter
__device__ double getLimiter(double r) {

    // // Minbee
    // if (r <= 0) {return 0.0;}
    // if (r > 0 && r <= 1) {return r;}
    // if (r > 1) {return fmin(1.0, 2.0 / (1 + r));}

    // Superbee
    if (r <= 0.0) {return 0.0;}
    if (r > 0.0 and r <= 0.5) {double res = 2 * r; return res;}
    if (r > 0.5 and r <= 1.0) {return 1.0;}
    if (r > 1.0) {double temp = fmin(r, 2.0 / (1 + r)); return fmin(temp, 2.0);}

    return 0;
}


// function: data reconstruction for a single cell
__device__ void dataReconstruct(double* u_backward, double* u_forward, double* u) {

    for (int v = 0; v < NUM_VARS; v++) {
        double q0 = u_backward[v], q = u[v], q1 = u_forward[v];

        double r = (q - q0) / (q1 - q);
        double slope_limiter = getLimiter(r);
        // double slope_limiter = 0.0;

        double delta_backward = q - q0;
        double delta_forward = q1 - q;
        double delta_i = 0.5 * (delta_backward + delta_forward);

        double qBarBackward = q - 0.5 * slope_limiter * delta_i;
        double qBarForward = q + 0.5 * slope_limiter * delta_i;

        u_backward[v] = qBarBackward;
        u_forward[v] = qBarForward;
    }
}


// function: calculate flux functions
template<int axis>
__device__ void flux_func(double* flux, const double* u_cons) {

    double u_prim[NUM_VARS];
    cons2primDevice(u_prim, u_cons);
    double rho = u_cons[0], momx = u_cons[1], momy = u_cons[2], E = u_cons[3];
    double vx = u_prim[1], vy = u_prim[2], p = u_prim[3];

    double rho_flux = axis == 0 ? momx : momy;
    double momx_flux = axis == 0 ? rho * pow(vx, 2) + p : rho * vx * vy;
    double momy_flux = axis == 0 ? rho * vy * vx : rho * pow(vy, 2) + p;
    double E_flux = axis == 0 ? (E + p) * vx : (E + p) * vy;

    flux[0] = rho_flux;
    flux[1] = momx_flux;
    flux[2] = momy_flux;
    flux[3] = E_flux;
}


// function: half-time step update
template<int axis>
__device__ void halfTimeStepUpdate(double* u_backward, double* u_forward, const double dx, const double dy, const double dt) {

    // calculate flux functions
    double flux_f[NUM_VARS], flux_b[NUM_VARS];
    flux_func<axis>(flux_b, u_backward);
    flux_func<axis>(flux_f, u_forward);

    // update
    double unit_len = axis == 0 ? dx : dy;
    for (int v = 0; v < NUM_VARS; v++) {
        double flux_update = 0.5 * (dt / unit_len) * (flux_f[v] - flux_b[v]);
        u_backward[v] = u_backward[v] - flux_update;
        u_forward[v] = u_forward[v] - flux_update;
    }
}


// function: calculate numerical fluxes with FORCE scheme
template<int axis>
__device__ void calFlux(double* flux, double* u_backward, double* u_forward, const double dx, const double dy, const double dt) {

    // calculate flux functions
    double flux_f[NUM_VARS], flux_b[NUM_VARS];
    flux_func<axis>(flux_b, u_backward);
    flux_func<axis>(flux_f, u_forward);

    // L-F scheme
    double unit_len = axis == 0 ? dx : dy;
    const double F_rho_LF = 0.5 * unit_len / dt * (u_backward[0] - u_forward[0]) + 0.5 * (flux_b[0] + flux_f[0]);
    const double F_momx_LF = 0.5 * unit_len / dt * (u_backward[1] - u_forward[1]) + 0.5 * (flux_b[1] + flux_f[1]);
    const double F_momy_LF = 0.5 * unit_len / dt * (u_backward[2] - u_forward[2]) + 0.5 * (flux_b[2] + flux_f[2]);
    const double F_E_LF = 0.5 * unit_len / dt * (u_backward[3] - u_forward[3]) + 0.5 * (flux_b[3] + flux_f[3]);

    // RI scheme
    double u_half_cons[NUM_VARS]; double F_RI[NUM_VARS];
    u_half_cons[0] = 0.5 * (u_backward[0] + u_forward[0]) - 0.5 * dt / unit_len * (flux_f[0] - flux_b[0]);
    u_half_cons[1] = 0.5 * (u_backward[1] + u_forward[1]) - 0.5 * dt / unit_len * (flux_f[1] - flux_b[1]);
    u_half_cons[2] = 0.5 * (u_backward[2] + u_forward[2]) - 0.5 * dt / unit_len * (flux_f[2] - flux_b[2]);
    u_half_cons[3] = 0.5 * (u_backward[3] + u_forward[3]) - 0.5 * dt / unit_len * (flux_f[3] - flux_b[3]);
    flux_func<axis>(F_RI, u_half_cons);

    // FORCE scheme
    const double F_rho_FORCE = 0.5 * (F_rho_LF + F_RI[0]);
    const double F_momx_FORCE = 0.5 * (F_momx_LF + F_RI[1]);
    const double F_momy_FORCE = 0.5 * (F_momy_LF + F_RI[2]);
    const double F_E_FORCE = 0.5 * (F_E_LF + F_RI[3]);
    flux[0] = F_rho_FORCE;
    flux[1] = F_momx_FORCE;
    flux[2] = F_momy_FORCE;
    flux[3] = F_E_FORCE;
}


// function: SLIC evolution in a single kernel with shared memory optimization
__global__ void SLIC_Preparation_X(Grid uBarL, Grid uBarR, Grid u, const double dx, const double dy, const double dt) {

    double uL[NUM_VARS];
    double uI[NUM_VARS];
    double uR[NUM_VARS];

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // data reconstruction and half-time update
    int i_min = nGhost - 1;
    int i_max = u.nCellsX + nGhost + 1;
    int j_min = nGhost;
    int j_max = u.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {

        // read data from global memory
        for (int v = 0; v < NUM_VARS; v++) {
            uI[v] = u(i, j, v);
            uL[v] = u(i - 1, j, v);
            uR[v] = u(i + 1, j, v);
        }

        // data reconstruction
        dataReconstruct(uL, uR, uI);
        // half time-step update
        halfTimeStepUpdate<0>(uL, uR, dx, dy, dt);

        // store updated data
        for (int v = 0; v < NUM_VARS; v++) {
            uBarL(i, j, v) = uL[v];
            uBarR(i, j, v) = uR[v];
        }
    }
}


__global__ void SLIC_CalFlux_X(Grid Flux, Grid uBarL, Grid uBarR, const double dx, const double dy, const double dt) {

    double cur_flux[NUM_VARS];
    double uL[NUM_VARS];
    double uR[NUM_VARS];

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // get data
    for (int v = 0; v < NUM_VARS; v++) {
        uL[v] = uBarR(i, j, v);
        uR[v] = uBarL(i + 1, j, v);
    }

    // calculate fluxes
    int i_min = nGhost - 1;
    int i_max = uBarL.nCellsX + nGhost;
    int j_min = nGhost;
    int j_max = uBarL.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {
        calFlux<0>(cur_flux, uL, uR, dx, dy, dt);
        // store fluxes
        for (int v = 0; v < NUM_VARS; v++) {
            Flux(i, j, v) = cur_flux[v];
        }
    }
}


__global__ void SLIC_Update_X(Grid u, Grid Flux, const double dx, const double dt) {

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // evolution by adding fluxes
    int i_min = nGhost;
    int i_max = u.nCellsX + nGhost;
    int j_min = nGhost;
    int j_max = u.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {
        for (int v = 0; v < NUM_VARS; v++) {
            u(i, j, v) = u(i, j, v) - dt / dx * (Flux(i, j, v) - Flux(i - 1, j, v));
        }
    }
}


__global__ void SLIC_Preparation_Y(Grid uBarL, Grid uBarR, Grid u, const double dx, const double dy, const double dt) {

    double uL[NUM_VARS];
    double uI[NUM_VARS];
    double uR[NUM_VARS];

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // data reconstruction and half-time update
    int i_min = nGhost;
    int i_max = u.nCellsX + nGhost;
    int j_min = nGhost - 1;
    int j_max = u.nCellsY + nGhost + 1;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {

        // read data from global memory
        for (int v = 0; v < NUM_VARS; v++) {
            uI[v] = u(i, j, v);
            uL[v] = u(i, j - 1, v);
            uR[v] = u(i, j + 1, v);
        }

        // data reconstruction
        dataReconstruct(uL, uR, uI);
        // half time-step update
        halfTimeStepUpdate<1>(uL, uR, dx, dy, dt);

        // store updated data
        for (int v = 0; v < NUM_VARS; v++) {
            uBarL(i, j, v) = uL[v];
            uBarR(i, j, v) = uR[v];
        }
    }
}


__global__ void SLIC_CalFlux_Y(Grid Flux, Grid uBarL, Grid uBarR, const double dx, const double dy, const double dt) {

    double cur_flux[NUM_VARS];
    double uL[NUM_VARS];
    double uR[NUM_VARS];

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // get data
    for (int v = 0; v < NUM_VARS; v++) {
        uL[v] = uBarR(i, j, v);
        uR[v] = uBarL(i, j + 1, v);
    }

    // calculate fluxes
    int i_min = nGhost;
    int i_max = uBarL.nCellsX + nGhost;
    int j_min = nGhost - 1;
    int j_max = uBarL.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {
        calFlux<1>(cur_flux, uL, uR, dx, dy, dt);
        // store fluxes
        for (int v = 0; v < NUM_VARS; v++) {
            Flux(i, j, v) = cur_flux[v];
        }
    }
}


__global__ void SLIC_Update_Y(Grid u, Grid Flux, const double dy, const double dt) {

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // evolution by adding fluxes
    int i_min = nGhost;
    int i_max = u.nCellsX + nGhost;
    int j_min = nGhost;
    int j_max = u.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {
        for (int v = 0; v < NUM_VARS; v++) {
            u(i, j, v) = u(i, j, v) - dt / dy * (Flux(i, j, v) - Flux(i, j - 1, v));
        }
    }
}


// function: data recording
void dataRecord(Grid uHost, Grid u, const int case_id, const double nCellsX, const double nCellsY,
    const double x0, const double y0, const double dx, const double dy, const double t) {

    // copy data from GPU to CPU
    hipMemcpy(uHost.data, u.data, (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost) * NUM_VARS * sizeof(double),
        hipMemcpyDeviceToHost);

    // check whether the directory exists, create one if not
    std::ostringstream folderPath;
    folderPath << "D:/Study_Master/WrittenAssignment/WorkSpace_CUDA/GPU/res/Case_" << case_id;
    std::string caseFolder = folderPath.str();
    if (!fs::exists(caseFolder)) {
        fs::create_directories(caseFolder);
    }

    // data recording
    std::ostringstream oss;
    oss << caseFolder << "/T=" << std::setprecision(2) << t << ".txt";
    std::string fileName = oss.str();
    std::fstream outFile(fileName, std::ios::out);

    double* u_ij_prim = new double[NUM_VARS];
    double* u_ij_cons = new double[NUM_VARS];

    for (int i = nGhost; i < nCellsX + nGhost; i++) {
        for (int j = nGhost; j < nCellsY + nGhost; j++) {
            for (int v = 0; v < NUM_VARS; v++) {
                u_ij_cons[v] = uHost(i, j, v);
            }
            cons2primHost(u_ij_prim, u_ij_cons);

            outFile << x0 + (i - nGhost + 0.5) * dx << ", " << y0 + (j - nGhost + 0.5) * dy
            << ", " << u_ij_prim[0] << ", " << u_ij_prim[1] << ", " << u_ij_prim[2] << ", " << u_ij_prim[3]
            << std::endl;
        }
    }
    outFile.close();
}


// function: initialization
__global__ void initialize(Grid u, const double x0, const double y0, const double y1,
    const double dx, const double dy, const int case_id) {

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    double bubble_center_x = 35;
    double bubble_center_y = 0.5 * y1;
    double bubble_radius = 25;

    if (i >= nGhost && i < u.nCellsX + nGhost && j >= nGhost && j < u.nCellsY + nGhost) {
        double x = x0 + (i - nGhost + 0.5) * dx;
        double y = y0 + (j - nGhost + 0.5) * dy;
        double u_ij_prim[NUM_VARS];
        double u_ij_cons[NUM_VARS];

        if (case_id == 1) {
            if (x >= 0.5 && y >= 0.5) {
                u_ij_prim[0] = 1.5;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 1.5;
            }
            if (x < 0.5 && y >= 0.5) {
                u_ij_prim[0] = 0.5325;
                u_ij_prim[1] = 1.206;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 0.3;
            }
            if (x < 0.5 && y < 0.5) {
                u_ij_prim[0] = 0.138;
                u_ij_prim[1] = 1.206;
                u_ij_prim[2] = 1.206;
                u_ij_prim[3] = 0.029;
            }
            if (x >= 0.5 && y < 0.5) {
                u_ij_prim[0] = 0.5325;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 1.206;
                u_ij_prim[3] = 0.3;
            }
        }

        if (case_id == 2) {
            if (x < 5) {
                u_ij_prim[0] = 1.7755;
                u_ij_prim[1] = 110.63;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 159060.0;
            }
            else if (pow(pow(x - bubble_center_x, 2) + pow(y - bubble_center_y, 2), 0.5) <= bubble_radius) {
                u_ij_prim[0] = 0.214;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 101325.0;
            }
            else {
                u_ij_prim[0] = 1.29;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 101325.0;
            }
        }

        // transform from primitive to conservative and store
        prim2consDevice(u_ij_cons, u_ij_prim);
        u(i, j, 0) = u_ij_cons[0];
        u(i, j, 1) = u_ij_cons[1];
        u(i, j, 2) = u_ij_cons[2];
        u(i, j, 3) = u_ij_cons[3];
    }
}


// function: mainloop
int main() {

    // experimental options
    int case_id = 2;  // Case 1: Quadrant problem; Case 2: Shock-Bubble interaction
    bool Record = false;  // whether record experimental data
    bool optTime = true;  // whether optimize dt calculation with shared memory

    // parameters
    std::array<int, 2> nCellsX_list = {400, 500};
    std::array<int, 2> nCellsY_list = {400, 197};
    std::array<double, 2> x1_list = {1.0, 225.0};
    std::array<double, 2> y1_list = {1.0, 89.0};
    std::array<double, 2> tStop_list = {0.3, 0.3};

    double x0 = 0.0, y0 = 0.0, tStart = 0.0;
    int nCellsX = nCellsX_list[case_id - 1], nCellsY = nCellsY_list[case_id - 1];
    double x1 = x1_list[case_id - 1], y1 = y1_list[case_id - 1];
    double dx = (x1 - x0) / nCellsX, dy = (y1 - y0) / nCellsY;
    double tStop = tStop_list[case_id - 1];

    // initialization and boundary conditions
    int nBlocksX = (nCellsX + 2 * nGhost + nThreadsX - 1) / nThreadsX;
    int nBlocksY = (nCellsY + 2 * nGhost + nThreadsY - 1) / nThreadsY;
    dim3 dimBlock(nThreadsX, nThreadsY, 1);
    dim3 dimGrid(nBlocksX, nBlocksY, 1);

    // x-direction evolution (without overlapping)
    int nBlocksXSLICX = (nCellsX + 2 * nGhost + nThreadsXSLICX - 1) / nThreadsXSLICX;
    int nBlocksYSLICX = (nCellsY + 2 * nGhost + nThreadsYSLICX - 1) / nThreadsYSLICX;
    dim3 dimBlockSLICX(nThreadsXSLICX, nThreadsYSLICX, 1);
    dim3 dimGridSLICX(nBlocksXSLICX, nBlocksYSLICX, 1);

    // y-direction evolution (without overlapping)
    int nBlocksXSLICY = (nCellsX + 2 * nGhost + nThreadsXSLICY - 1) / nThreadsXSLICY;
    int nBlocksYSLICY = (nCellsY + 2 * nGhost + nThreadsYSLICY - 1) / nThreadsYSLICY;
    dim3 dimBlockSLICY(nThreadsXSLICY, nThreadsYSLICY, 1);
    dim3 dimGridSLICY(nBlocksXSLICY, nBlocksYSLICY, 1);

    // execution time recording
    double elapsdt = 0, elapsx = 0, elapsy = 0, elapsbc = 0, elapstotal = 0;
    clock_t startx, endx, starty, endy, startdt, enddt, startbc, endbc, start, end;

    // initialization
    Grid uHost(nCellsX, nCellsY, CPU);  // data on CPU for recording
    Grid u(nCellsX, nCellsY, GPU);  // data in conservative form on GPU
    Grid uBarL(nCellsX, nCellsY, GPU);  // data after half-time evolution on GPU
    Grid uBarR(nCellsX, nCellsY, GPU);  // data after half-time evolution on GPU
    Grid Flux(nCellsX, nCellsY, GPU);  // fluxes on GPU

    initialize<<<dimGrid, dimBlock>>>(u, x0, y0, y1, dx, dy, case_id);
    CUDA_CHECK;

    // boundary conditions
    setBoundaryCondition<<<dimGrid, dimBlock>>>(u);
    CUDA_CHECK;

    // update data
    double t = tStart;
    std::array<double, 3> t_record_list = {0.1, 0.2, 0.3};
    int record_index = 0;
    int counter = 0;
    do {
        start = clock();

        // compute time step
        startdt = clock();
        double dt = computeTimeStep(u, dx, dy, dimGrid, dimBlock, optTime);
        enddt = clock();
        elapsdt += (double)(enddt - startdt) / CLOCKS_PER_SEC;

        t = t + dt;
        counter++;
        std::cout << "ite = " << counter<< ", time = " << t << std::endl;

        // x-direction evolution
        startx = clock();
        SLIC_Preparation_X<<<dimGridSLICX, dimBlockSLICX>>>(uBarL, uBarR, u, dx, dy, dt);
        CUDA_CHECK;
        SLIC_CalFlux_X<<<dimGridSLICX, dimBlockSLICX>>>(Flux, uBarL, uBarR, dx, dy, dt);
        CUDA_CHECK;
        SLIC_Update_X<<<dimGridSLICX, dimBlockSLICX>>>(u, Flux, dx, dt);
        CUDA_CHECK;
        endx = clock();
        elapsx += (double)(endx - startx) / CLOCKS_PER_SEC;

        // boundary conditions
        startbc = clock();
        setBoundaryCondition<<<dimGrid, dimBlock>>>(u);
        CUDA_CHECK;
        endbc = clock();
        elapsbc += (double)(endbc - startbc) / CLOCKS_PER_SEC;

        // y-direction evolution
        starty = clock();
        SLIC_Preparation_Y<<<dimGridSLICY, dimBlockSLICY>>>(uBarL, uBarR, u, dx, dy, dt);
        CUDA_CHECK;
        SLIC_CalFlux_Y<<<dimGridSLICY, dimBlockSLICY>>>(Flux, uBarL, uBarR, dx, dy, dt);
        CUDA_CHECK;
        SLIC_Update_Y<<<dimGridSLICY, dimBlockSLICY>>>(u, Flux, dy, dt);
        CUDA_CHECK;
        endy = clock();
        elapsy += (double)(endy - starty) / CLOCKS_PER_SEC;

        // boundary conditions
        startbc = clock();
        setBoundaryCondition<<<dimGrid, dimBlock>>>(u);
        CUDA_CHECK;
        endbc = clock();
        elapsbc += (double)(endbc - startbc) / CLOCKS_PER_SEC;

        end = clock();
        elapstotal += (double)(end - start) / CLOCKS_PER_SEC;

        // data recording
        if (Record && t >= t_record_list[record_index]) {
            std::cout << "Recording: t = " << t << std::endl;
            dataRecord(uHost, u, case_id, nCellsX, nCellsY, x0, y0, dx, dy, t);
            record_index++;
        }

    } while (t < tStop);

    // release memory
    hipFree(u.data);
    hipFree(uBarL.data);
    hipFree(uBarR.data);
    hipFree(Flux.data);
    delete[] uHost.data;

    // output time recording
    std::cout << "=== Timing Results ===" << std::endl;
    std::cout << "Total execution time: " << elapstotal << " sec" << std::endl;
    std::cout << "computeTimeStep: " << elapsdt << " sec" << std::endl;
    std::cout << "Boundary Conditions: " << elapsbc << " sec" << std::endl;
    std::cout << "X-direction evolution: " << elapsx << " sec" << std::endl;
    std::cout << "Y-direction evolution: " << elapsy << " sec" << std::endl;

    return 0;
}

