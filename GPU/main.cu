//
// Created by Lenovo on 25-03-01.
//

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <array>

// macro: constant parameters
#define NUM_VARS 4  // number of independent variables
#define nGhost 2  // number of ghost cells in each direction
#define C 0.8  // CFL number
#define gamma 1.4  // adiabatic index of ideal gas
#define nThreadsX 32  // number of threads per block in x-direction
#define nThreadsY 32  // number of threads per block in y-direction
#define nThreadsXOverlap 6  // number of threads per block in x-direction in SLIC
#define nThreadsYOverlap 6  // number of threads per block in y-direction in SLIC

// macro: debugging
#define CUDA_CHECK {\
    hipDeviceSynchronize();\
    hipError_t err = hipGetLastError();\
    if(err){\
        std::cout << "Error: " << hipGetErrorString(err) << " line " << __LINE__ << std::endl;\
        exit(1);\
    }\
}


// structure: data storage and process
struct Grid {
    double *data;
    int nCellsX, nCellsY;
    double x0, x1, y0, y1;

    // format for building an instance
    Grid(const int nX, const int nY, const std::array<double, 4>& sim_range) {
        nCellsX = nX;
        nCellsY = nY;
        x0 = sim_range[0], x1 = sim_range[1];
        y0 = sim_range[2], y1 = sim_range[3];
        hipMalloc((void **)& data, (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost) * NUM_VARS * sizeof(double));
    }

    // operator functions
    __device__ __host__
    double operator() (const int i, const int j, const int v) const {
        return data[i + j * (nCellsX + 2 * nGhost) + v * (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost)];
    }
    __device__ __host__
    double& operator() (const int i, const int j, const int v) {
        return data[i + j * (nCellsX + 2 * nGhost) + v * (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost)];
    }
};


// function: transform from primitive to conservative on CPU
std::array<double, 4> prim2consHost(std::array<double, 4> const& u_ij) {

    const double rho = u_ij[0];
    const double u = u_ij[1];
    const double v = u_ij[2];
    const double p = u_ij[3];

    std::array<double, 4> res{};
    res[0] = rho;  // rho
    res[1] = rho * u;  // momx
    res[2] = rho * v;  // momy
    res[3] = p / (gamma - 1) + 0.5 * rho * (pow(u, 2) + pow(v, 2));  // E

    return res;
}


// function: transform from primitive to conservative on GPU
__device__ void prim2consDevice(double* u_ij_cons, const double* u_ij_prim) {

    const double rho = u_ij_prim[0];
    const double u = u_ij_prim[1];
    const double v = u_ij_prim[2];
    const double p = u_ij_prim[3];

    u_ij_cons[0] = rho;  // rho
    u_ij_cons[1] = rho * u;  // momx
    u_ij_cons[2] = rho * v;  // momy
    u_ij_cons[3] = p / (gamma - 1) + 0.5 * rho * (pow(u, 2) + pow(v, 2));  // E
}


// function: transform from conservative to primitive on CPU
std::array<double, 4> cons2primHost(std::array<double, 4> const& u_ij) {

    const double rho = u_ij[0];
    const double momx = u_ij[1];
    const double momy = u_ij[2];
    const double E = u_ij[3];

    std::array<double, 4> res{};
    res[0] = rho;  // rho
    res[1] = momx / rho;  // u
    res[2] = momy / rho;  // v
    res[3] = (gamma - 1) * (E - 0.5 * pow(momx, 2) / rho - 0.5 * pow(momy, 2) / rho);  // p

    return res;
}


// function: transform from conservative to primitive on GPU
__device__ void cons2primDevice(double* u_ij_prim, const double* u_ij_cons) {

    const double rho = u_ij_cons[0];
    const double momx = u_ij_cons[1];
    const double momy = u_ij_cons[2];
    const double E = u_ij_cons[3];

    u_ij_prim[0] = rho;  // rho
    u_ij_prim[1] = momx / rho;  // u
    u_ij_prim[2] = momy / rho;  // v
    u_ij_prim[3] = (gamma - 1) * (E - 0.5 * pow(momx, 2) / rho - 0.5 * pow(momy, 2) / rho);  // p
}


// function: set transmissive boundary conditions
__global__ void setBoundaryCondition(Grid u) {

    const int nCellsX = u.nCellsX, nCellsY = u.nCellsY;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // transmissive boundary condition
    for (int v = 0; v < NUM_VARS; v++) {
        // lower boundary
        if (i >= nGhost && i < nCellsX + nGhost && j >= 0 && j < nGhost) {
            u(i, j, v) = u(i, nGhost, v);
        }
        // upper boundary
        if (i >= nGhost && i < nCellsX + nGhost && j >= nCellsY + nGhost && j < nCellsY + 2 * nGhost) {
            u(i, j, v) = u(i, nCellsY + nGhost - 1, v);
        }
        // left boundary
        if (j >= 0 && j < nCellsY + 2 * nGhost && i >= 0 && i < nGhost) {
            u(i, j, v) = u(nGhost, j, v);
        }
        // right boundary
        if (j >= 0 && j < nCellsY + 2 * nGhost && i >= nCellsX + nGhost && i < nCellsX + 2 * nGhost) {
            u(i, j, v) = u(nCellsX + nGhost - 1, j, v);
        }
    }
}


// function: calculate the maximum velocity in the whole grid on GPU
__global__ void computeAmax(double* aDevice, Grid u) {

    // shared memory for current block
    __shared__ double aBlock[nThreadsX][nThreadsY];

    // variable substitution
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i_local = threadIdx.x, j_local = threadIdx.y;
    double cur_v = 0.0;

    // calculate a's in current block
    if (i >= nGhost && i < u.nCellsX + nGhost && j >= nGhost && j < u.nCellsY + nGhost) {
        double u_prim[NUM_VARS];
        double u_cons[NUM_VARS];
        for (int v = 0; v < NUM_VARS; v++) {
            u_cons[v] = u(i, j, v);
        }
        cons2primDevice(u_prim, u_cons);
        double cur_vx = u_prim[1], cur_vy = u_prim[2];
        cur_v = pow(pow(cur_vx, 2) + pow(cur_vy, 2), 0.5);
    }
    aBlock[threadIdx.x][threadIdx.y] = cur_v;
    __syncthreads();

    // block-wise reduction
    if (blockDim.x >= 32 && blockDim.y >= 32 && i_local < 16 && j_local < 16) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 16]),
            fmax(aBlock[i_local + 16][j_local], aBlock[i_local + 16][j_local + 16]));
        __syncthreads();
    }
    if (blockDim.x >= 16 && blockDim.y >= 16 && i_local < 8 && j_local < 8) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 8]),
            fmax(aBlock[i_local + 8][j_local], aBlock[i_local + 8][j_local + 8]));
        __syncthreads();
    }
    if (blockDim.x >= 8 && blockDim.y >= 8 && i_local < 4 && j_local < 4) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 4]),
            fmax(aBlock[i_local + 4][j_local], aBlock[i_local + 4][j_local + 4]));
        __syncthreads();
    }
    if (blockDim.x >= 4 && blockDim.y >= 4 && i_local < 2 && j_local < 2) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 2]),
            fmax(aBlock[i_local + 2][j_local], aBlock[i_local + 2][j_local + 2]));
        __syncthreads();
    }
    if (blockDim.x >= 2 && blockDim.y >= 2 && i_local < 1 && j_local < 1) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 1]),
            fmax(aBlock[i_local + 1][j_local], aBlock[i_local + 1][j_local + 1]));
        __syncthreads();
    }

    // a_max in current block
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        aDevice[blockIdx.y * gridDim.x + blockIdx.x] = aBlock[threadIdx.x][threadIdx.y];
    }
}


// function: calculate time step
double computeTimeStep(const Grid& u, const double& dx, const double& dy, const dim3& dimGrid, const dim3& dimBlock) {

    // calculate a_max on GPU
    double* aDevice;
    int a_size = dimGrid.x * dimGrid.y;
    hipMalloc(&aDevice, a_size * sizeof(double));
    computeAmax<<<dimGrid, dimBlock>>>(aDevice, u);
    CUDA_CHECK;

    // transfer data to CPU
    double* aHost = new double [a_size];
    hipMemcpy(aHost, aDevice, a_size * sizeof(double), hipMemcpyDeviceToHost);

    // for stability: numerical dependence stencil should contain the largest wave speed
    double a_max = 0.0;
    for (int i = 0; i < a_size; i++) {
        a_max = std::max(a_max, aHost[i]);
    }
    hipFree(aDevice);
    delete[] aHost;
    const double timeStep = C * std::min(dx, dy) / a_max;
    return timeStep;
}


// function: initialization
__global__ void initialize(Grid u, const double x0, const double y0, const double y1,
    const double dx, const double dy, const int case_id) {

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    double bubble_center_x = 35;
    double bubble_center_y = 0.5 * y1;
    double bubble_radius = 25;

    if (i >= nGhost && i < u.nCellsX + nGhost && j >= nGhost && j < u.nCellsY + nGhost) {
        double x = x0 + (i - nGhost + 0.5) * dx;
        double y = y0 + (j - nGhost + 0.5) * dy;
        double u_ij_prim[NUM_VARS];
        double u_ij_cons[NUM_VARS];

        if (case_id == 1) {
            if (x >= 0.5 && y >= 0.5) {
                u_ij_prim[0] = 1.5;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 1.5;
            }
            if (x < 0.5 && y >= 0.5) {
                u_ij_prim[0] = 0.5325;
                u_ij_prim[1] = 1.206;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 0.3;
            }
            if (x < 0.5 && y < 0.5) {
                u_ij_prim[0] = 0.138;
                u_ij_prim[1] = 1.206;
                u_ij_prim[2] = 1.206;
                u_ij_prim[3] = 0.029;
            }
            if (x >= 0.5 && y < 0.5) {
                u_ij_prim[0] = 0.5325;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 1.206;
                u_ij_prim[3] = 0.3;
            }
        }

        if (case_id == 2) {
            if (x < 5) {
                u_ij_prim[0] = 1.7755;
                u_ij_prim[1] = 110.63;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 159060.0;
            }
            else if (pow(pow(x - bubble_center_x, 2) + pow(y - bubble_center_y, 2), 0.5) <= bubble_radius) {
                u_ij_prim[0] = 0.214;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 101325.0;
            }
            else {
                u_ij_prim[0] = 1.29;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 101325.0;
            }
        }

        // transform from primitive to conservative and store
        prim2consDevice(u_ij_cons, u_ij_prim);
        u(i, j, 0) = u_ij_cons[0];
        u(i, j, 1) = u_ij_cons[1];
        u(i, j, 2) = u_ij_cons[2];
        u(i, j, 3) = u_ij_cons[3];
    }
}


// function: mainloop
int main() {

    // parameters
    int case_id = 1;
    std::array<int, 2> nCellsX_list = {400, 500};
    std::array<int, 2> nCellsY_list = {400, 197};
    std::array<double, 2> x1_list = {1.0, 225.0};
    std::array<double, 2> y1_list = {1.0, 89.0};
    std::array<double, 2> tStop_list = {0.3, 0.3};

    double x0 = 0.0, y0 = 0.0, tStart = 0.0;
    int nCellsX = nCellsX_list[case_id - 1], nCellsY = nCellsY_list[case_id - 1];
    double x1 = x1_list[case_id - 1], y1 = y1_list[case_id - 1];
    double dx = (x1 - x0) / nCellsX, dy = (y1 - y0) / nCellsY;
    double tStop = tStop_list[case_id - 1];

    // initialization
    std::array<double, 4> sim_range = {x0, x1, y0, y1};
    Grid u(nCellsX, nCellsY, sim_range);  // in conservative form

    int nBlocksX = (int)ceil((nCellsX + 2 * nGhost) / nThreadsX);
    int nBlocksY = (int)ceil((nCellsY + 2 * nGhost) / nThreadsY);
    dim3 dimBlock(nThreadsX, nThreadsY, 1);
    dim3 dimGrid(nBlocksX, nBlocksY, 1);

    initialize<<<dimGrid, dimBlock>>>(u, x0, y0, y1, dx, dy, case_id);
    CUDA_CHECK;

    // boundary conditions
    setBoundaryCondition<<<dimGrid, dimBlock>>>(u);
    CUDA_CHECK;

    // update data
    double t = tStart;
    // std::array t_record_list = {0.1, 0.2, 0.3};
    // int record_index = 0;
    int counter = 0;
    do {
        // compute time step
        double dt = computeTimeStep(u, dx, dy, dimGrid, dimBlock);
        t = t + dt;
        std::cout << "ite = " << counter + 1 << ", time = " << t << std::endl;

    } while (t < tStop);

    return 0;
}

