#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <fstream>
#include <cassert>
#include <filesystem>
#include <ctime>

namespace fs = std::filesystem;

// macro: constant parameters
#define NUM_VARS 4  // number of independent variables
#define nGhost 2  // number of ghost cells in each direction
#define C 0.8  // CFL number
#define gamma 1.4  // adiabatic index of ideal gas
#define nThreadsX 32  // number of threads per block in x-direction
#define nThreadsY 32  // number of threads per block in y-direction
#define nThreadsXSLICX 32  // number of threads per block in x-direction for SLIC evolution in x-direction
#define nThreadsYSLICX 4  // number of threads per block in y-direction for SLIC evolution in x-direction
#define nThreadsXSLICY 4  // number of threads per block in x-direction for SLIC evolution in y-direction
#define nThreadsYSLICY 32  // number of threads per block in y-direction for SLIC evolution in y-direction

// macro: debugging
#define CUDA_CHECK {\
    hipDeviceSynchronize();\
    hipError_t err = hipGetLastError();\
    if(err){\
        std::cout << "Error: " << hipGetErrorString(err) << " line " << __LINE__ << std::endl;\
        exit(1);\
    }\
}

enum Processor {CPU, GPU};

// structure: data storage and process
struct Grid {

    // attributes
    double* data;
    int nCellsX;
    int nCellsY;
    Processor processor;

    // format for building an instance
    Grid(const int nX, const int nY, Processor p) {
        nCellsX = nX;
        nCellsY = nY;
        processor = p;
        switch (processor) {
            case CPU:
                data = new double[(nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost) * NUM_VARS];
                break;
            case GPU:
                hipMalloc((void **)& data, (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost) * NUM_VARS * sizeof(double));
                break;
        }
    }

    // operator functions
    __device__ __host__
    double operator() (const int i, const int j, const int v) const {
        return data[i + j * (nCellsX + 2 * nGhost) + v * (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost)];
    }
    __device__ __host__
    double& operator() (const int i, const int j, const int v) {
        return data[i + j * (nCellsX + 2 * nGhost) + v * (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost)];
    }
};


// function: transform from primitive to conservative on CPU
void prim2consHost(double* u_ij_cons, const double* u_ij_prim) {

    const double rho = u_ij_prim[0];
    const double u = u_ij_prim[1];
    const double v = u_ij_prim[2];
    const double p = u_ij_prim[3];

    u_ij_cons[0] = rho;  // rho
    u_ij_cons[1] = rho * u;  // momx
    u_ij_cons[2] = rho * v;  // momy
    u_ij_cons[3] = p / (gamma - 1) + 0.5 * rho * (pow(u, 2) + pow(v, 2));  // E
}


// function: transform from primitive to conservative on GPU
__device__ void prim2consDevice(double* u_ij_cons, const double* u_ij_prim) {

    const double rho = u_ij_prim[0];
    const double u = u_ij_prim[1];
    const double v = u_ij_prim[2];
    const double p = u_ij_prim[3];

    u_ij_cons[0] = rho;  // rho
    u_ij_cons[1] = rho * u;  // momx
    u_ij_cons[2] = rho * v;  // momy
    u_ij_cons[3] = p / (gamma - 1) + 0.5 * rho * (pow(u, 2) + pow(v, 2));  // E
}


// function: transform from conservative to primitive on CPU
void cons2primHost(double* u_ij_prim, const double* u_ij_cons) {

    const double rho = u_ij_cons[0];
    const double momx = u_ij_cons[1];
    const double momy = u_ij_cons[2];
    const double E = u_ij_cons[3];

    u_ij_prim[0] = rho;  // rho
    u_ij_prim[1] = momx / rho;  // u
    u_ij_prim[2] = momy / rho;  // v
    u_ij_prim[3] = (gamma - 1) * (E - 0.5 * pow(momx, 2) / rho - 0.5 * pow(momy, 2) / rho);  // p
}


// function: transform from conservative to primitive on GPU
__device__ void cons2primDevice(double* u_ij_prim, const double* u_ij_cons) {

    const double rho = u_ij_cons[0];
    const double momx = u_ij_cons[1];
    const double momy = u_ij_cons[2];
    const double E = u_ij_cons[3];

    u_ij_prim[0] = rho;  // rho
    u_ij_prim[1] = momx / rho;  // u
    u_ij_prim[2] = momy / rho;  // v
    u_ij_prim[3] = (gamma - 1) * (E - 0.5 * pow(momx, 2) / rho - 0.5 * pow(momy, 2) / rho);  // p
}


// function: set transmissive boundary conditions
__global__ void setBoundaryCondition(Grid u) {

    const int nCellsX = u.nCellsX, nCellsY = u.nCellsY;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // transmissive boundary condition
    for (int v = 0; v < NUM_VARS; v++) {
        // lower boundary
        if (i >= nGhost && i < nCellsX + nGhost && j >= 0 && j < nGhost) {
            u(i, j, v) = u(i, nGhost, v);
        }
        // upper boundary
        if (i >= nGhost && i < nCellsX + nGhost && j >= nCellsY + nGhost && j < nCellsY + 2 * nGhost) {
            u(i, j, v) = u(i, nCellsY + nGhost - 1, v);
        }
        // left boundary
        if (j >= 0 && j < nCellsY + 2 * nGhost && i >= 0 && i < nGhost) {
            u(i, j, v) = u(nGhost, j, v);
        }
        // right boundary
        if (j >= 0 && j < nCellsY + 2 * nGhost && i >= nCellsX + nGhost && i < nCellsX + 2 * nGhost) {
            u(i, j, v) = u(nCellsX + nGhost - 1, j, v);
        }
    }
}


// function: calculate the maximum velocity in each block on GPU
__global__ void computeAmaxOpt(double* aDevice, Grid u) {

    // shared memory for current block
    __shared__ double aBlock[nThreadsX][nThreadsY];

    // variable substitution
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i_local = threadIdx.x, j_local = threadIdx.y;
    double cur_a = 0.0;

    // calculate a's in current block
    if (i >= nGhost && i < u.nCellsX + nGhost && j >= nGhost && j < u.nCellsY + nGhost) {
        double u_prim[NUM_VARS];
        double u_cons[NUM_VARS];
        for (int v = 0; v < NUM_VARS; v++) {
            u_cons[v] = u(i, j, v);
        }
        cons2primDevice(u_prim, u_cons);
        double cur_rho = u_prim[0], cur_vx = u_prim[1], cur_vy = u_prim[2], cur_p = u_prim[3];
        double cur_v = pow(pow(cur_vx, 2) + pow(cur_vy, 2), 0.5);
        double Cs = pow(gamma * cur_p / cur_rho, 0.5);
        cur_a = cur_v + Cs;
    }
    aBlock[threadIdx.x][threadIdx.y] = cur_a;
    __syncthreads();

    // block-wise reduction
    if (blockDim.x >= 32 && blockDim.y >= 32 && i_local < 16 && j_local < 16) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 16]),
            fmax(aBlock[i_local + 16][j_local], aBlock[i_local + 16][j_local + 16]));
        __syncthreads();
    }
    if (blockDim.x >= 16 && blockDim.y >= 16 && i_local < 8 && j_local < 8) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 8]),
            fmax(aBlock[i_local + 8][j_local], aBlock[i_local + 8][j_local + 8]));
        __syncthreads();
    }
    if (blockDim.x >= 8 && blockDim.y >= 8 && i_local < 4 && j_local < 4) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 4]),
            fmax(aBlock[i_local + 4][j_local], aBlock[i_local + 4][j_local + 4]));
        __syncthreads();
    }
    if (blockDim.x >= 4 && blockDim.y >= 4 && i_local < 2 && j_local < 2) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 2]),
            fmax(aBlock[i_local + 2][j_local], aBlock[i_local + 2][j_local + 2]));
        __syncthreads();
    }
    if (blockDim.x >= 2 && blockDim.y >= 2 && i_local < 1 && j_local < 1) {
        aBlock[i_local][j_local] = fmax(fmax(aBlock[i_local][j_local], aBlock[i_local][j_local + 1]),
            fmax(aBlock[i_local + 1][j_local], aBlock[i_local + 1][j_local + 1]));
        __syncthreads();
    }

    // a_max in current block
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        aDevice[blockIdx.y * gridDim.x + blockIdx.x] = aBlock[threadIdx.x][threadIdx.y];
    }
}


// function: calculate the velocity in each cell on GPU
__global__ void computeAmax(double* aDevice, Grid u) {

    // variable substitution
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    double cur_a = 0.0;

    // calculate a
    if (i >= nGhost && i < u.nCellsX + nGhost && j >= nGhost && j < u.nCellsY + nGhost) {
        double u_prim[NUM_VARS];
        double u_cons[NUM_VARS];
        for (int v = 0; v < NUM_VARS; v++) {
            u_cons[v] = u(i, j, v);
        }
        cons2primDevice(u_prim, u_cons);
        double cur_rho = u_prim[0], cur_vx = u_prim[1], cur_vy = u_prim[2], cur_p = u_prim[3];
        double cur_v = pow(pow(cur_vx, 2) + pow(cur_vy, 2), 0.5);
        double Cs = pow(gamma * cur_p / cur_rho, 0.5);
        cur_a = cur_v + Cs;
    }

    aDevice[j * blockDim.x * gridDim.x + i] = cur_a;
}


// function: calculate time step
double computeTimeStep(Grid u, const double& dx, const double& dy, const dim3& dimGrid, const dim3& dimBlock, bool optTime) {

    // calculate a's on GPU
    double* aDevice;
    int a_size = 0;
    if (optTime) {  // with shared memory optimization
        a_size = dimGrid.x * dimGrid.y;
        hipMalloc(&aDevice, a_size * sizeof(double));
        computeAmaxOpt<<<dimGrid, dimBlock>>>(aDevice, u);
        CUDA_CHECK;
    } else {  // without shared memory optimization
        a_size = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y;
        hipMalloc(&aDevice, a_size * sizeof(double));
        computeAmax<<<dimGrid, dimBlock>>>(aDevice, u);
        CUDA_CHECK;
    }

    // transfer data to CPU
    double* aHost = new double [a_size];
    hipMemcpy(aHost, aDevice, a_size * sizeof(double), hipMemcpyDeviceToHost);

    // for stability: numerical dependence stencil should contain the largest wave speed
    double a_max = 0.0;
    for (int i = 0; i < a_size; i++) {
        a_max = std::max(a_max, aHost[i]);
    }
    hipFree(aDevice);
    delete[] aHost;
    const double timeStep = C * std::min(dx, dy) / a_max;
    return timeStep;
}


// function: calculate slope limiter
__device__ double getLimiter(double r) {

    // // Minbee
    // if (r <= 0) {return 0.0;}
    // if (r > 0 && r <= 1) {return r;}
    // if (r > 1) {return fmin(1.0, 2.0 / (1 + r));}

    // Superbee
    if (r <= 0.0) {return 0.0;}
    if (r > 0.0 and r <= 0.5) {double res = 2 * r; return res;}
    if (r > 0.5 and r <= 1.0) {return 1.0;}
    if (r > 1.0) {double temp = fmin(r, 2.0 / (1 + r)); return fmin(temp, 2.0);}

    return 0;
}


// function: data reconstruction for a single cell
__device__ void dataReconstruct(double* u_backward, double* u_forward, double* u) {

    for (int v = 0; v < NUM_VARS; v++) {
        double q0 = u_backward[v], q = u[v], q1 = u_forward[v];

        double r = (q - q0) / (q1 - q);
        double slope_limiter = getLimiter(r);
        // double slope_limiter = 0.0;

        double delta_backward = q - q0;
        double delta_forward = q1 - q;
        double delta_i = 0.5 * (delta_backward + delta_forward);

        double qBarBackward = q - 0.5 * slope_limiter * delta_i;
        double qBarForward = q + 0.5 * slope_limiter * delta_i;

        u_backward[v] = qBarBackward;
        u_forward[v] = qBarForward;
    }
}


// function: calculate flux functions
template<int axis>
__device__ void flux_func(double* flux, const double* u_cons) {

    double u_prim[NUM_VARS];
    cons2primDevice(u_prim, u_cons);
    double rho = u_cons[0], momx = u_cons[1], momy = u_cons[2], E = u_cons[3];
    double vx = u_prim[1], vy = u_prim[2], p = u_prim[3];

    double rho_flux = axis == 0 ? momx : momy;
    double momx_flux = axis == 0 ? rho * pow(vx, 2) + p : rho * vx * vy;
    double momy_flux = axis == 0 ? rho * vy * vx : rho * pow(vy, 2) + p;
    double E_flux = axis == 0 ? (E + p) * vx : (E + p) * vy;

    flux[0] = rho_flux;
    flux[1] = momx_flux;
    flux[2] = momy_flux;
    flux[3] = E_flux;
}


// function: half-time step update
template<int axis>
__device__ void halfTimeStepUpdate(double* u_backward, double* u_forward, const double dx, const double dy, const double dt) {

    // calculate flux functions
    double flux_f[NUM_VARS], flux_b[NUM_VARS];
    flux_func<axis>(flux_b, u_backward);
    flux_func<axis>(flux_f, u_forward);

    // update
    double unit_len = axis == 0 ? dx : dy;
    for (int v = 0; v < NUM_VARS; v++) {
        double flux_update = 0.5 * (dt / unit_len) * (flux_f[v] - flux_b[v]);
        u_backward[v] = u_backward[v] - flux_update;
        u_forward[v] = u_forward[v] - flux_update;
    }
}


// function: calculate numerical fluxes with FORCE scheme
template<int axis>
__device__ void calFlux(double* flux, double* u_backward, double* u_forward, const double dx, const double dy, const double dt) {

    // calculate flux functions
    double flux_f[NUM_VARS], flux_b[NUM_VARS];
    flux_func<axis>(flux_b, u_backward);
    flux_func<axis>(flux_f, u_forward);

    // L-F scheme
    double unit_len = axis == 0 ? dx : dy;
    const double F_rho_LF = 0.5 * unit_len / dt * (u_backward[0] - u_forward[0]) + 0.5 * (flux_b[0] + flux_f[0]);
    const double F_momx_LF = 0.5 * unit_len / dt * (u_backward[1] - u_forward[1]) + 0.5 * (flux_b[1] + flux_f[1]);
    const double F_momy_LF = 0.5 * unit_len / dt * (u_backward[2] - u_forward[2]) + 0.5 * (flux_b[2] + flux_f[2]);
    const double F_E_LF = 0.5 * unit_len / dt * (u_backward[3] - u_forward[3]) + 0.5 * (flux_b[3] + flux_f[3]);

    // RI scheme
    double u_half_cons[NUM_VARS]; double F_RI[NUM_VARS];
    u_half_cons[0] = 0.5 * (u_backward[0] + u_forward[0]) - 0.5 * dt / unit_len * (flux_f[0] - flux_b[0]);
    u_half_cons[1] = 0.5 * (u_backward[1] + u_forward[1]) - 0.5 * dt / unit_len * (flux_f[1] - flux_b[1]);
    u_half_cons[2] = 0.5 * (u_backward[2] + u_forward[2]) - 0.5 * dt / unit_len * (flux_f[2] - flux_b[2]);
    u_half_cons[3] = 0.5 * (u_backward[3] + u_forward[3]) - 0.5 * dt / unit_len * (flux_f[3] - flux_b[3]);
    flux_func<axis>(F_RI, u_half_cons);

    // FORCE scheme
    const double F_rho_FORCE = 0.5 * (F_rho_LF + F_RI[0]);
    const double F_momx_FORCE = 0.5 * (F_momx_LF + F_RI[1]);
    const double F_momy_FORCE = 0.5 * (F_momy_LF + F_RI[2]);
    const double F_E_FORCE = 0.5 * (F_E_LF + F_RI[3]);
    flux[0] = F_rho_FORCE;
    flux[1] = F_momx_FORCE;
    flux[2] = F_momy_FORCE;
    flux[3] = F_E_FORCE;
}


// function: SLIC evolution in a single kernel with shared memory optimization
__global__ void SLIC_Evolution_X(Grid u, const double dx, const double dy, const double dt) {

    // allocate shared memory
    __shared__ double uL[nThreadsXSLICX][nThreadsYSLICX][NUM_VARS];
    __shared__ double uI[nThreadsXSLICX][nThreadsYSLICX][NUM_VARS];
    __shared__ double uR[nThreadsXSLICX][nThreadsYSLICX][NUM_VARS];
    __shared__ double flux[nThreadsXSLICX][nThreadsYSLICX][NUM_VARS];

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x - 2 * blockIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i_local = threadIdx.x, j_local = threadIdx.y;
    if (i_local >= nThreadsXSLICX || j_local >= nThreadsYSLICX) {assert(false);}

    // data reconstruction and half-time update
    int i_min = nGhost - 1;
    int i_max = u.nCellsX + nGhost + 1;
    int j_min = nGhost;
    int j_max = u.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {

        // read data from global memory
        for (int v = 0; v < NUM_VARS; v++) {
            uI[i_local][j_local][v] = u(i, j, v);
        }
        __syncthreads();

        // read uL and uR
        for (int v = 0; v < NUM_VARS; v++) {
            if (i_local == 0) {
                uL[i_local][j_local][v] = u(i - 1, j, v);
            } else {
                uL[i_local][j_local][v] = uI[i_local - 1][j_local][v];
            }
            if (i_local == blockDim.x - 1) {
                uR[i_local][j_local][v] = u(i + 1, j, v);
            } else {
                uR[i_local][j_local][v] = uI[i_local + 1][j_local][v];
            }
        }

        // data reconstruction
        dataReconstruct(uL[i_local][j_local], uR[i_local][j_local], uI[i_local][j_local]);
        // half time-step update
        halfTimeStepUpdate<0>(uL[i_local][j_local], uR[i_local][j_local], dx, dy, dt);
        __syncthreads();
    }

    // calculate fluxes
    i_min = nGhost - 1;
    i_max = u.nCellsX + nGhost;
    j_min = nGhost;
    j_max = u.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {
        if (i_local < blockDim.x - 1) {
            calFlux<0>(flux[i_local][j_local], uR[i_local][j_local], uL[i_local + 1][j_local], dx, dy, dt);
        }
        __syncthreads();
    }

    // evolution by adding fluxes
    i_min = nGhost;
    i_max = u.nCellsX + nGhost;
    j_min = nGhost;
    j_max = u.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {
        if (i_local > 0 && i_local < blockDim.x - 1) {
            for (int v = 0; v < NUM_VARS; v++) {
                u(i, j, v) = u(i, j, v) - dt / dx * (flux[i_local][j_local][v] - flux[i_local - 1][j_local][v]);
            }
        }
        __syncthreads();
    }
}


__global__ void SLIC_Evolution_Y(Grid u, const double dx, const double dy, const double dt) {

    // allocate shared memory
    __shared__ double uL[nThreadsXSLICY][nThreadsYSLICY][NUM_VARS];
    __shared__ double uI[nThreadsXSLICY][nThreadsYSLICY][NUM_VARS];
    __shared__ double uR[nThreadsXSLICY][nThreadsYSLICY][NUM_VARS];
    __shared__ double flux[nThreadsXSLICY][nThreadsYSLICY][NUM_VARS];

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y - 2 * blockIdx.y;
    int i_local = threadIdx.x, j_local = threadIdx.y;

    // data reconstruction and half-time update
    int i_min = nGhost;
    int i_max = u.nCellsX + nGhost;
    int j_min = nGhost - 1;
    int j_max = u.nCellsY + nGhost + 1;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {

        // read data from global memory
        for (int v = 0; v < NUM_VARS; v++) {
            uI[i_local][j_local][v] = u(i, j, v);
        }
        __syncthreads();

        // read uL and uR
        for (int v = 0; v < NUM_VARS; v++) {
            if (j_local == 0) {
                uL[i_local][j_local][v] = u(i, j - 1, v);
            } else {
                uL[i_local][j_local][v] = uI[i_local][j_local - 1][v];
            }
            if (j_local == blockDim.y - 1) {
                uR[i_local][j_local][v] = u(i, j + 1, v);
            } else {
                uR[i_local][j_local][v] = uI[i_local][j_local + 1][v];
            }
        }

        // data reconstruction
        dataReconstruct(uL[i_local][j_local], uR[i_local][j_local], uI[i_local][j_local]);
        // half time-step update
        halfTimeStepUpdate<1>(uL[i_local][j_local], uR[i_local][j_local], dx, dy, dt);
        __syncthreads();
    }

    // calculate fluxes
    i_min = nGhost;
    i_max = u.nCellsX + nGhost;
    j_min = nGhost - 1;
    j_max = u.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {
        if (j_local < blockDim.y - 1) {
            calFlux<1>(flux[i_local][j_local], uR[i_local][j_local], uL[i_local][j_local + 1], dx, dy, dt);
        }
        __syncthreads();
    }

    // evolution by adding fluxes
    i_min = nGhost;
    i_max = u.nCellsX + nGhost;
    j_min = nGhost;
    j_max = u.nCellsY + nGhost;
    if (i >= i_min && i < i_max && j >= j_min && j < j_max) {
        if (j_local > 0 && j_local < blockDim.y - 1) {
            for (int v = 0; v < NUM_VARS; v++) {
                u(i, j, v) = u(i, j, v) - dt / dy * (flux[i_local][j_local][v] - flux[i_local][j_local - 1][v]);
            }
        }
        __syncthreads();
    }
}


// function: data recording
void dataRecord(Grid uHost, Grid u, const int case_id, const double nCellsX, const double nCellsY,
    const double x0, const double y0, const double dx, const double dy, const double t) {

    // copy data from GPU to CPU
    hipMemcpy(uHost.data, u.data, (nCellsX + 2 * nGhost) * (nCellsY + 2 * nGhost) * NUM_VARS * sizeof(double),
        hipMemcpyDeviceToHost);

    // check whether the directory exists, create one if not
    std::ostringstream folderPath;
    folderPath << "res/Case_" << case_id;
    std::string caseFolder = folderPath.str();
    if (!fs::exists(caseFolder)) {
        fs::create_directories(caseFolder);
    }

    // data recording
    std::ostringstream oss;
    oss << caseFolder << "/T=" << std::setprecision(2) << t << ".txt";
    std::string fileName = oss.str();
    std::fstream outFile(fileName, std::ios::out);

    double* u_ij_prim = new double[NUM_VARS];
    double* u_ij_cons = new double[NUM_VARS];

    for (int i = nGhost; i < nCellsX + nGhost; i++) {
        for (int j = nGhost; j < nCellsY + nGhost; j++) {
            for (int v = 0; v < NUM_VARS; v++) {
                u_ij_cons[v] = uHost(i, j, v);
            }
            cons2primHost(u_ij_prim, u_ij_cons);

            outFile << x0 + (i - nGhost + 0.5) * dx << ", " << y0 + (j - nGhost + 0.5) * dy
            << ", " << u_ij_prim[0] << ", " << u_ij_prim[1] << ", " << u_ij_prim[2] << ", " << u_ij_prim[3]
            << std::endl;
        }
    }
    outFile.close();
}


// function: initialization
__global__ void initialize(Grid u, const double x0, const double y0, const double dx, const double dy,
    const int case_id, const double bubble_center_x, double bubble_center_y, double bubble_radius) {

    // get coordinates
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= nGhost && i < u.nCellsX + nGhost && j >= nGhost && j < u.nCellsY + nGhost) {
        double x = x0 + (i - nGhost + 0.5) * dx;
        double y = y0 + (j - nGhost + 0.5) * dy;
        double u_ij_prim[NUM_VARS];
        double u_ij_cons[NUM_VARS];

        if (case_id == 1) {
            if (x >= 0.5 && y >= 0.5) {
                u_ij_prim[0] = 1.5;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 1.5;
            }
            if (x < 0.5 && y >= 0.5) {
                u_ij_prim[0] = 0.5325;
                u_ij_prim[1] = 1.206;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 0.3;
            }
            if (x < 0.5 && y < 0.5) {
                u_ij_prim[0] = 0.138;
                u_ij_prim[1] = 1.206;
                u_ij_prim[2] = 1.206;
                u_ij_prim[3] = 0.029;
            }
            if (x >= 0.5 && y < 0.5) {
                u_ij_prim[0] = 0.5325;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 1.206;
                u_ij_prim[3] = 0.3;
            }
        }

        if (case_id == 2) {
            if (x < 0.005) {
                u_ij_prim[0] = 1.7755;
                u_ij_prim[1] = 110.63;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 159060.0;
            }
            else if (pow(pow(x - bubble_center_x, 2) + pow(y - bubble_center_y, 2), 0.5) <= bubble_radius) {
                u_ij_prim[0] = 0.214;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 101325.0;
            }
            else {
                u_ij_prim[0] = 1.29;
                u_ij_prim[1] = 0.0;
                u_ij_prim[2] = 0.0;
                u_ij_prim[3] = 101325.0;
            }
        }

        // transform from primitive to conservative and store
        prim2consDevice(u_ij_cons, u_ij_prim);
        u(i, j, 0) = u_ij_cons[0];
        u(i, j, 1) = u_ij_cons[1];
        u(i, j, 2) = u_ij_cons[2];
        u(i, j, 3) = u_ij_cons[3];
    }
}


// function: check memory usage
void checkKernelAttributes() {
    hipFuncAttributes attr;

    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(computeAmaxOpt));
    std::cout << "=== Kernel Resource Usage: computeAmaxOpt ===" << std::endl;
    std::cout << "Registers used: " << attr.numRegs << std::endl;
    std::cout << "Shared memory per block: " << attr.sharedSizeBytes << " bytes" << std::endl;
    std::cout << "Constant memory used: " << attr.constSizeBytes << " bytes" << std::endl;
    std::cout << "Local memory per thread: " << attr.localSizeBytes << " bytes" << std::endl;
    std::cout << "Max threads per block: " << attr.maxThreadsPerBlock << std::endl;
    std::cout << std::endl;

    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(SLIC_Evolution_X));
    std::cout << "=== Kernel Resource Usage: SLIC_Evolution_X ===" << std::endl;
    std::cout << "Registers used: " << attr.numRegs << std::endl;
    std::cout << "Shared memory per block: " << attr.sharedSizeBytes << " bytes" << std::endl;
    std::cout << "Constant memory used: " << attr.constSizeBytes << " bytes" << std::endl;
    std::cout << "Local memory per thread: " << attr.localSizeBytes << " bytes" << std::endl;
    std::cout << "Max threads per block: " << attr.maxThreadsPerBlock << std::endl;
    std::cout << std::endl;

    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(SLIC_Evolution_Y));
    std::cout << "=== Kernel Resource Usage: SLIC_Evolution_Y ===" << std::endl;
    std::cout << "Registers used: " << attr.numRegs << std::endl;
    std::cout << "Shared memory per block: " << attr.sharedSizeBytes << " bytes" << std::endl;
    std::cout << "Constant memory used: " << attr.constSizeBytes << " bytes" << std::endl;
    std::cout << "Local memory per thread: " << attr.localSizeBytes << " bytes" << std::endl;
    std::cout << "Max threads per block: " << attr.maxThreadsPerBlock << std::endl;
    std::cout << std::endl;
}


// function: mainloop
int main() {

    // experimental options
    int case_id = 2;  // Case 1: Quadrant problem; Case 2: Shock-Bubble interaction
    bool Record = false;  // whether record experimental data
    bool optTime = true;  // whether optimize dt calculation with shared memory
    double bubble_center_x = 0.035, bubble_center_y = 0.0445, bubble_radius = 0.025;
    double Ms = 1.22, p_Air = 101325.0, rho_Air = 1.29;
    double Cs_Air = pow(gamma * p_Air / rho_Air, 0.5);
    double time_ratio = bubble_radius / (Cs_Air * Ms);
    if (case_id == 1) {time_ratio = 1;}

    // parameters
    std::array<int, 2> nCellsX_list = {400, 500};
    std::array<int, 2> nCellsY_list = {400, 197};
    std::array<double, 2> x1_list = {1.0, 0.225};
    std::array<double, 2> y1_list = {1.0, 0.089};
    std::array<double, 2> tStop_list = {0.3, 7.8 * time_ratio};

    double x0 = 0.0, y0 = 0.0, tStart = 0.0;
    int nCellsX = nCellsX_list[case_id - 1], nCellsY = nCellsY_list[case_id - 1];
    double x1 = x1_list[case_id - 1], y1 = y1_list[case_id - 1];
    double dx = (x1 - x0) / nCellsX, dy = (y1 - y0) / nCellsY;
    double tStop = tStop_list[case_id - 1];

    int nBlocksX = (nCellsX + 2 * nGhost + nThreadsX - 1) / nThreadsX;
    int nBlocksY = (nCellsY + 2 * nGhost + nThreadsY - 1) / nThreadsY;
    dim3 dimBlock(nThreadsX, nThreadsY, 1);
    dim3 dimGrid(nBlocksX, nBlocksY, 1);

    // x-direction evolution with overlapping blocks
    int nBlocksXSLICX = (nCellsX + 2 * nGhost - nThreadsXSLICX + nThreadsXSLICX - 3) / (nThreadsXSLICX - 2) + 1;
    int nBlocksYSLICX = (nCellsY + 2 * nGhost + nThreadsYSLICX - 1) / nThreadsYSLICX;
    dim3 dimBlockSLICX(nThreadsXSLICX, nThreadsYSLICX, 1);
    dim3 dimGridSLICX(nBlocksXSLICX, nBlocksYSLICX, 1);

    // y-direction evolution with overlapping blocks
    int nBlocksXSLICY = (nCellsX + 2 * nGhost + nThreadsXSLICY - 1) / nThreadsXSLICY;
    int nBlocksYSLICY = (nCellsY + 2 * nGhost - nThreadsYSLICY + nThreadsYSLICY - 3) / (nThreadsYSLICY - 2) + 1;
    dim3 dimBlockSLICY(nThreadsXSLICY, nThreadsYSLICY, 1);
    dim3 dimGridSLICY(nBlocksXSLICY, nBlocksYSLICY, 1);

    // execution time recording
    double elapsdt = 0, elapsx = 0, elapsy = 0, elapsbc = 0, elapstotal = 0;
    clock_t startx, endx, starty, endy, startdt, enddt, startbc, endbc, start, end;

    // initialization
    Grid uHost(nCellsX, nCellsY, CPU);  // data on CPU for recording
    Grid u(nCellsX, nCellsY, GPU);  // data in conservative form on GPU
    initialize<<<dimGrid, dimBlock>>>(u, x0, y0, dx, dy, case_id, bubble_center_x, bubble_center_y, bubble_radius);
    CUDA_CHECK;

    // boundary conditions
    setBoundaryCondition<<<dimGrid, dimBlock>>>(u);
    CUDA_CHECK;

    // update data
    double t = tStart;
    int counter = 0;
    do {
        start = clock();
        // check memory usage
        if (counter == 0) {checkKernelAttributes();}

        // compute time step
        startdt = clock();
        double dt = computeTimeStep(u, dx, dy, dimGrid, dimBlock, optTime);
        enddt = clock();
        elapsdt += (double)(enddt - startdt) / CLOCKS_PER_SEC;

        t = t + dt;
        counter++;
        std::cout << "ite = " << counter<< ", time = " << t << std::endl;

        // x-direction evolution
        startx = clock();
        SLIC_Evolution_X<<<dimGridSLICX, dimBlockSLICX>>>(u, dx, dy, dt);
        CUDA_CHECK;
        endx = clock();
        elapsx += (double)(endx - startx) / CLOCKS_PER_SEC;

        // boundary conditions
        startbc = clock();
        setBoundaryCondition<<<dimGrid, dimBlock>>>(u);
        CUDA_CHECK;
        endbc = clock();
        elapsbc += (double)(endbc - startbc) / CLOCKS_PER_SEC;

        // y-direction evolution
        starty = clock();
        SLIC_Evolution_Y<<<dimGridSLICY, dimBlockSLICY>>>(u, dx, dy, dt);
        CUDA_CHECK;
        endy = clock();
        elapsy += (double)(endy - starty) / CLOCKS_PER_SEC;

        // boundary conditions
        startbc = clock();
        setBoundaryCondition<<<dimGrid, dimBlock>>>(u);
        CUDA_CHECK;
        endbc = clock();
        elapsbc += (double)(endbc - startbc) / CLOCKS_PER_SEC;

        end = clock();
        elapstotal += (double)(end - start) / CLOCKS_PER_SEC;

    } while (t < tStop);

    // data recording
    if (Record) {
        std::cout << "Recording: t = " << t << std::endl;
        dataRecord(uHost, u, case_id, nCellsX, nCellsY, x0, y0, dx, dy, t / time_ratio);
    }

    // release memory
    hipFree(u.data);
    delete[] uHost.data;

    // output time recording
    std::cout << "=== Timing Results ===" << std::endl;
    std::cout << "Total execution time: " << elapstotal << " sec" << std::endl;
    std::cout << "computeTimeStep: " << elapsdt << " sec" << std::endl;
    std::cout << "Boundary Conditions: " << elapsbc << " sec" << std::endl;
    std::cout << "X-direction evolution: " << elapsx << " sec" << std::endl;
    std::cout << "Y-direction evolution: " << elapsy << " sec" << std::endl;

    return 0;
}

