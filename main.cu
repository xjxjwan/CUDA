#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


int main() {
    
    // parameters
    double C = 0.8;
    double gama = 1.4;
    int case_id = 2;
    int nCellsX = 0;
    int nCellsY = 0;
    double tStart = 0.0;
    double tStop = 0.0;
    double x0 = 0.0, y0 = 0.0;
    double x1 = 0.0, y1 = 0.0;
    double dx = 0.0, dy = 0.0;
    std::vector<std::vector<std::array<double, 4>>> u{};

    // initial data
    if (case_id == 1) { // Quadrant problem

        nCellsX = 400; nCellsY = 400;
        x1 = 1.0; y1 = 1.0;
        dx = (x1 - x0) / nCellsX;
        dy = (y1 - y0) / nCellsY;
        tStop = 0.3;
        u.resize(nCellsX + 4, std::vector<std::array<double, 4>>(nCellsY + 4));  // 4 ghost cells

        for (int i = 2; i < nCellsX + 2; i++) {
            for (int j = 2; j < nCellsY + 2; j++) {

                // get coordinates
                double x = x0 + (i - 1.5) * dx;
                double y = y0 + (j - 1.5) * dy;
                std::array<double, 4> u_ij{};

                if (x >= 0.5 && y >= 0.5) {u_ij = {1.5, 0.0, 0.0, 1.5};}
                if (x < 0.5 && y >= 0.5) {u_ij = {0.5325, 1.206, 0.0, 0.3};}
                if (x < 0.5 && y < 0.5) {u_ij = {0.138, 1.206, 1.206, 0.029};}
                if (x >= 0.5 && y < 0.5) {u_ij = {0.5325, 0.0, 1.206, 0.3};}

                // transform from primitive to conservative
                u[i][j] = prim2cons(u_ij, gama);
            }
        }
    }

    if (case_id == 2) { // Shock-bubble interaction

        nCellsX = 500; nCellsY = 197;
        x1 = 225; y1 = 89;
        double bubble_center_x = 35;
        double bubble_center_y = 0.5 * y1;
        dx = (x1 - x0) / nCellsX;
        dy = (y1 - y0) / nCellsY;
        tStop = 0.3;
        u.resize(nCellsX + 4, std::vector<std::array<double, 4>>(nCellsY + 4));  // 4 ghost cells

        for (int i = 2; i < nCellsX + 2; i++) {
            for (int j = 2; j < nCellsY + 2; j++) {

                // get coordinates
                double x = x0 + (i - 1.5) * dx;
                double y = y0 + (j - 1.5) * dy;
                std::array<double, 4> u_ij{};

                if (x < 5) {  // air left to shock
                    u_ij = {1.7755, 110.63, 0.0, 159060.0};
                } else if (pow(pow(x - bubble_center_x, 2) + pow(y - bubble_center_y, 2), 0.5) <= 25) {  // inside bubble
                    u_ij = {0.214, 0.0, 0.0, 101325.0};
                } else {  // air right to shock
                    u_ij = {1.29, 0.0, 0.0, 101325.0};
                }

                // transform from primitive to conservative
                u[i][j] = prim2cons(u_ij, gama);
            }
        }
    }

    // boundary condition
    setBoundaryCondition(u, nCellsX, nCellsY);

}