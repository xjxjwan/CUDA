#include <iostream>
#include <hip/hip_runtime.h>


// CUDA Kernel：将数组的每个元素加 1
__global__ void addOneKernel(int *d_array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_array[idx] += 1;
    }
}


int main() {
    // 定义数组大小
    const int N = 10;
    int h_array[N];  // 主机端数组
    int *d_array;    // 设备端数组指针

    // 初始化主机数组
    for (int i = 0; i < N; i++) {
        h_array[i] = i;
    }

    // 申请设备端内存
    hipMalloc((void**)&d_array, N * sizeof(int));

    // 将数据从主机复制到设备
    hipMemcpy(d_array, h_array, N * sizeof(int), hipMemcpyHostToDevice);

    // 启动 Kernel，每个 block 10 个线程
    addOneKernel<<<1, 10>>>(d_array, N);

    // 将结果从设备复制回主机
    hipMemcpy(h_array, d_array, N * sizeof(int), hipMemcpyDeviceToHost);

    // 打印结果
    std::cout << "Results after Kernel execution: ";
    for (int i = 0; i < N; i++) {
        std::cout << h_array[i] << " ";
    }
    std::cout << std::endl;

    // 释放设备内存
    hipFree(d_array);

    return 0;
}

